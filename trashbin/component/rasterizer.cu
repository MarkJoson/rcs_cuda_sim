#include "hip/hip_runtime.h"
#include <vector>
#include <glog/logging.h>
#include <ranges>
#include <algorithm>
#include <iostream>
#include <hipcub/hipcub.hpp>
#include <thrust/device_vector.h>
#include <boost/range/join.hpp>
#include <SFML/Graphics.hpp>
#include <glm/glm.hpp>
#include "mapgen/mapgen_generate.h"
#include "mapgen/mapgen_postprocess.h"

union Line{
    struct {
        float sx;
        float sy;
        float ex;
        float ey;
    };
    float4 data;
};


std::vector<Line> lines;

constexpr int MAP_WIDTH = 80;
constexpr int MAP_HEIGHT = 60;
constexpr int GRID_SIZE = 1;

constexpr float SCALING = 8;
constexpr float OFFSET = 20;

constexpr float PI = M_PI;

#define LIDAR_LINES_LOG2    (10)         // 128lines
#define LIDAR_LINES         (1<<LIDAR_LINES_LOG2)

#define TILE_SIZE_LOG2      (5)         // 32lines
#define TILE_SIZE           (1<<TILE_SIZE_LOG2)
#define TILE_NUM            (1 << (LIDAR_LINES_LOG2-TILE_SIZE_LOG2))

#define CTA_SIZE            (128)
#define RASTER_WARPS        (CTA_SIZE/32)

#define LINE_BUF_SIZE       (CTA_SIZE*2)
#define FR_BUF_SIZE         (CTA_SIZE*2)
#define FRAG_BUF_SIZE       (CTA_SIZE*6)

#define EMIT_PER_THREAD     (2)
#define TOTAL_EMITION       (EMIT_PER_THREAD * CTA_SIZE)


#ifdef __DRIVER_TYPES_H__
static inline const char *_cudaGetErrorEnum(hipError_t error) {
  return hipGetErrorName(error);
}
#endif

template <typename T>
void check(T result, char const *const func, const char *const file,
           int const line) {
  if (result) {
    fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line,
            static_cast<unsigned int>(result), _cudaGetErrorEnum(result), func);
    throw std::exception();
    // exit(EXIT_FAILURE);
  }
}

#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)


// thrust::device_vector<bool> lidar_inst_state;
// thrust::device_vector<int> line_count;

struct RCSParam
{
    // int             *line_cnt;
    // float2          *line_s;
    // float2          *line_e;

    // float2          *pos;
    // bool            *inst_enabled;
    float           max_range;
    float           resolu_inv;
    float           resolu;
    int             ray_num;
};

RCSParam g_hparams {
    .max_range = 20,
    .resolu_inv = LIDAR_LINES / (2*M_PI),
    .resolu = (2*M_PI) / LIDAR_LINES,
    .ray_num = LIDAR_LINES
};


__constant__ __device__ RCSParam g_params;


__host__ __device__ bool prepareLine(float2 vs, float2 ve, float max_range)
{
    float cdot = vs.x*ve.y - vs.y*ve.x;
    float dx = ve.x-vs.x;
    float dy = ve.y-vs.y;
    // float len = sqrtf(dx*dx+dy*dy);
    float invsqrt = rsqrtf(dx*dx+dy*dy);
    float dist = fabs(cdot)*invsqrt;

    return dist < max_range && cdot>=0;
}

__forceinline__ __host__ __device__ float vec_atan2_0_360(float2 vec)
{
    float angle = atan2f(vec.y, vec.x);
    return angle < 0 ? angle + 2*PI : angle;
}

__forceinline__ __host__ __device__ float getR(float2 lb, float2 le, float theta)
{
    float sin,cos;
    sincosf(theta, &sin, &cos);
    float2 dvec = make_float2(le.x - lb.x, le.y - lb.y);
    return (lb.y*dvec.x-lb.x*dvec.y)/(dvec.x*sin-dvec.y*cos);
}

__forceinline__ __host__ __device__ uint32_t genMask(int start, int end)
{
    return ((1<<end)-1) & ~((1<<start)-1);
    // int pop = __builtin_popcount(mask);
}

// TODO. Texture 1D
__global__ void rasterKernel(
    int                         numLines,
    const float3 __restrict__*  poses,
    const float2 __restrict__*  line_begins,
    const float2 __restrict__*  line_ends,
             int             *  lidar_response
)
{
    using BlockScan = hipcub::BlockScan<uint32_t, CTA_SIZE>;
    using BlockRunLengthDecodeT = hipcub::BlockRunLengthDecode<uint32_t, CTA_SIZE, 1, EMIT_PER_THREAD>;

    volatile __shared__ uint32_t s_lineBuf[LINE_BUF_SIZE];           // 1K
    volatile __shared__ uint32_t s_frLineIdxBuf[FR_BUF_SIZE];        // 1K
    volatile __shared__ uint32_t s_frLineSGridFragsBuf[FR_BUF_SIZE]; // 1K
    __shared__ uint32_t s_lidarResponse[LIDAR_LINES];            // 1K
    __shared__ union {
        typename BlockScan::TempStorage scan_temp_storage;
        typename BlockRunLengthDecodeT::TempStorage decode_temp_storage;
    } temp_storage;

    uint32_t tid = threadIdx.x;
    uint32_t totalLineRead = 0;
    uint32_t lineBufRead=0, lineBufWrite=0;
    uint32_t frLineBufRead=0, frLineBufWrite=0;
    float3 pose = poses[blockIdx.x];

    /********* 初始化lidar数据 *********/
    for(int i=tid; i<g_params.ray_num; i+=CTA_SIZE)
        s_lidarResponse[i] = g_params.max_range*100;

    // if(tid == 0) printf("---------##### Task RECEIVED\n");

    for(;;)
    {
        /********* Load Lines to Buffer *********/

        while(lineBufWrite-lineBufRead < CTA_SIZE && totalLineRead < numLines)
        {
            // if(tid == 0) printf("[READ] LINE RANGE: %d~%d.\n", totalLineRead, totalLineRead+CTA_SIZE);
            uint32_t visibility = false;
            int lineIdx = totalLineRead+tid;
            if(lineIdx < numLines)
            {
                float2 lb = make_float2(line_begins[lineIdx].x-pose.x, line_begins[lineIdx].y-pose.y);
                float2 le = make_float2(line_ends[lineIdx].x-pose.x, line_ends[lineIdx].y-pose.y);
                visibility = prepareLine(lb, le, g_params.max_range);
                // if(visibility)
                // {
                //     const auto &vs=lb, &ve=le;
                //     float cdot = vs.x*ve.y - vs.y*ve.x;
                //     float dx = ve.x-vs.x;
                //     float dy = ve.y-vs.y;
                //     float len = sqrtf(dx*dx+dy*dy);
                //     float dist = fabs(cdot)/len;
                //     printf("[READ] PASS:%03d, \t[%.2f,%.2f]->[%.2f,%.2f], \t[%.2f,%.2f]->[%.2f,%.2f], \tarea:%.2f, \tdist%.2f\n", lineIdx,
                //         line_begins[lineIdx].x, line_begins[lineIdx].y, line_ends[lineIdx].x, line_ends[lineIdx].y,
                //         lb.x, lb.y, le.x, le.y, cdot, dist);
                // }
            }

            uint32_t scan, scan_reduce;
            BlockScan(temp_storage.scan_temp_storage).ExclusiveSum(visibility, scan, scan_reduce);

            if(visibility) {
                s_lineBuf[lineBufWrite+scan] = lineIdx;
            }

            lineBufWrite += scan_reduce;

            totalLineRead += CTA_SIZE;
            __syncthreads();
        }

        // 第二部分继续的条件：已经读取了128个，或没有读取128个，但没有新的线段

        // if(tid == 0) printf("[READ] FINISHED! TOTAL READ: %d, LINE BUF:%d\n", totalLineRead, lineBufWrite);

        /********* 计算终止栅格，细光栅化，存入待发射线段缓冲区 *********/
        // do
        {
            int lineIdx = -1;
            int frag = 0;
            int s_grid = -1;
            if(lineBufRead + tid < lineBufWrite)
            {
                lineIdx = s_lineBuf[(lineBufRead + tid) % LINE_BUF_SIZE];
                float2 lb = make_float2(line_begins[lineIdx].x-pose.x, line_begins[lineIdx].y-pose.y);
                float2 le = make_float2(line_ends[lineIdx].x-pose.x, line_ends[lineIdx].y-pose.y);

                auto s_angle = vec_atan2_0_360(lb);
                auto e_angle = vec_atan2_0_360(le);
                s_grid = s_angle * g_params.resolu_inv;
                int e_grid = e_angle * g_params.resolu_inv;

                frag = (e_grid-s_grid) + ((e_grid < s_grid) ? g_params.ray_num : 0);
                // printf("[RASTER] THREAD:%d, LINE_ID:%d, SGRID:%d, EGRID:%d, FRAG:%d\n", tid, lineIdx, s_grid, e_grid, frag);
            }

            // 压缩到FR_BUF队列中
            uint32_t scan, scan_sum;
            BlockScan(temp_storage.scan_temp_storage).ExclusiveSum(frag>0, scan, scan_sum);
            if(frag > 0)
            {
                uint32_t idx = (frLineBufWrite+scan) % FR_BUF_SIZE;
                s_frLineIdxBuf[idx] = lineIdx;
                s_frLineSGridFragsBuf[idx] = (s_grid << 16) | (frag & 0xffff);

            }
            frLineBufWrite += scan_sum;
            __syncthreads();

            //
            lineBufRead = min(lineBufRead+CTA_SIZE, lineBufWrite);
        }

        // 此时要么 读取了128个，要么 lineBuf处理完了
        // if(tid == 0) printf("[RASTER] FRBuf:[R:%d,W:%d] VALID: %d\n", frLineBufRead, frLineBufWrite, frLineBufWrite - frLineBufRead);

        // 第三部分继续的条件：读取到128个，或未读取到128个，但是已经无法再读取新的线段；
        if(frLineBufWrite-frLineBufRead < CTA_SIZE && (lineBufRead < lineBufWrite || totalLineRead < numLines)) {
            // if(tid == 0) printf("[RASTER] CONTINUE LOOP!\n");
            continue;
        }

        // if(tid == 0) printf("[RASTER] FINISHED!\n");

        /********* Count and Emit *********/
        do
        {
            // if(tid == 0) printf("[EMIT] LOAD LINE [%d-%d]\n", frLineBufRead, frLineBufWrite);
            // 加载CTA_SIZE个到缓冲区，准备进行Decode
            uint32_t runValue[1] = {0}, runLength[1] = {0};
            int frLineBufIdx = frLineBufRead + tid;
            if(frLineBufIdx < frLineBufWrite)
            {
                frLineBufIdx = frLineBufIdx % FR_BUF_SIZE;
                runValue[0] = frLineBufIdx;
                runLength[0] = s_frLineSGridFragsBuf[frLineBufIdx] & 0xffff;        // 取低16位的frag
            }
            frLineBufRead = min(frLineBufRead+CTA_SIZE, frLineBufWrite);
            __syncthreads();

            uint32_t total_decoded_size = 0;
            BlockRunLengthDecodeT blk_rld(temp_storage.decode_temp_storage, runValue, runLength, total_decoded_size);

            // 将本次读取的 CTA_SIZE*EMIT_PER_LINE 个frag全部发射
            uint32_t decoded_window_offset = 0;
            while(decoded_window_offset < total_decoded_size)
            {
                uint32_t relative_offsets[2];
                uint32_t decoded_items[2];
                uint32_t num_valid_items = min(total_decoded_size - decoded_window_offset, CTA_SIZE * EMIT_PER_THREAD);
                blk_rld.RunLengthDecode(decoded_items, relative_offsets, decoded_window_offset);
                decoded_window_offset += num_valid_items;

                #pragma unroll
                for(int i=0; i<2; i++)
                {
                    if(tid*EMIT_PER_THREAD + i >= num_valid_items)
                        break;

                    int fragIdx = relative_offsets[i];
                    uint32_t frLineBufIdx = decoded_items[i];
                    uint32_t lineIdx = s_frLineIdxBuf[frLineBufIdx];
                    int s_grid = s_frLineSGridFragsBuf[frLineBufIdx] >> 16;

                    float2 lb = make_float2(line_begins[lineIdx].x-pose.x, line_begins[lineIdx].y-pose.y);
                    float2 le = make_float2(line_ends[lineIdx].x-pose.x, line_ends[lineIdx].y-pose.y);
                    int grid = (s_grid + fragIdx + 1) % g_params.ray_num;
                    uint16_t response = getR(lb, le, grid*g_params.resolu) * 1024;        // 10位定点小数表示，最大距离64m
                    uint32_t resp_idx = response << 16 | lineIdx & 0xffff;
                    atomicMin_block(&s_lidarResponse[grid], resp_idx);
                }
            }
            __syncthreads();
        } while(frLineBufWrite != frLineBufRead && totalLineRead >= numLines);       // 继续的条件：已经没有办法读取更多的frag线段，则需要将剩余的frlineBufWrite处理完

        // if(tid == 0) printf("[EMIT] FINISHED!\n");

        // 全部线段已经处理完
        if(totalLineRead >= numLines) break;
    }

    for(int i=tid; i<g_params.ray_num; i+=CTA_SIZE)
        lidar_response[i] = s_lidarResponse[i];
}




std::vector<float> rasterGPU(int numLines, float3 pose, const std::vector<float2> &line_begins, const std::vector<float2> &line_ends)
{
    // float3 poses[1] {pose};
    thrust::device_vector<float3> poses(1);
    poses[0] = pose;

    thrust::device_vector<int> lidar_response(LIDAR_LINES);
    thrust::device_vector<float2> lbs = line_begins;
    thrust::device_vector<float2> les = line_ends;

    rasterKernel<<<1, CTA_SIZE>>>(numLines, poses.data().get(), lbs.data().get(), les.data().get(), lidar_response.data().get());
    checkCudaErrors(hipDeviceSynchronize());

    return std::vector<float>(lidar_response.begin(), lidar_response.end());
}

std::vector<float> rasterCPU(int numLines, float3 pose, const std::vector<float2> &line_begins, const std::vector<float2> &line_ends)
{
    // glm::vec2 pos = glm::vec2(inst_pos.x, inst_pos.y);

    // std::vector<int> lineFrag();
    // 初筛队列：背面剔除+距离剔除
    std::vector<int> lineBuf(LINE_BUF_SIZE);
    int lineBufRead=0, lineBufWrite=0;          // Read==Write空，(Write+1)%All==Read满

    // 交射线剔除
    std::vector<std::tuple<int, int, int>> frLineBuf(FR_BUF_SIZE);
    std::vector<int> frLineFrag(FR_BUF_SIZE);
    int frBufRead=0, frBufWrite=0;

    // 片段
    std::vector<std::pair<int,int>> fragBuf(CTA_SIZE*4+LIDAR_LINES);
    int fragRead=0, fragWrite=0;

    std::vector<float> response(LIDAR_LINES);
    // int totalFrag = 0;

    // TODO. 按线段长度排序

    for(int i=0; i<CTA_SIZE; i++)
        for(int j=i; j<g_hparams.ray_num; j+=CTA_SIZE)
            response[j] = g_hparams.max_range*100;

    // printf("---------##### Task RECEIVED\n");
    // 每次处理CTASize个线条
    for(int batchIdx=0; batchIdx<numLines; batchIdx+=CTA_SIZE)
    {
        // printf("[---★ NEW LOOP ★---] READ LINE FROM %d~%d.\n", batchIdx, batchIdx+CTA_SIZE);
        for(int i=0; i<CTA_SIZE; i++) {
            if(batchIdx + i >= numLines) {
                // printf("[READ] REACH TO THE END: %d.\n", batchIdx+i);
                break;
            }
            // 读取CTASize个线条，计算可见性，存入buf
            float2 lb = make_float2(line_begins[batchIdx+i].x-pose.x, line_begins[batchIdx+i].y-pose.y);
            float2 le = make_float2(line_ends[batchIdx+i].x-pose.x, line_ends[batchIdx+i].y-pose.y);
            bool visibility = prepareLine(lb, le, g_hparams.max_range);
            if(visibility) {        // TODO. compression 用Atomic还是用Exclusive Scan?
                lineBuf[lineBufWrite++ % LINE_BUF_SIZE] = batchIdx + i;
            }
        }

        // printf("[READ] TOTAL VISIBILITY: %d\n", lineBufWrite-lineBufRead);

        // 累计足够256个，或者剩余不足256个
        if((lineBufWrite - lineBufRead < CTA_SIZE) && (numLines - batchIdx >= CTA_SIZE))
        {
            // printf("[READ] Insufficient LINE: %d, Remain: %d, GO ON READING!\n", lineBufWrite-lineBufRead, numLines-batchIdx);
            continue;
        }

        // ---------------- 方案一：不分tile，针对每一条线段的占用格数计算前缀和发射数

        for(int i=0; i<CTA_SIZE; i++)
        {
            if (lineBufRead + i >= lineBufWrite)
                break;

            // 计算线程的数据索引
            int lineBufIdx = (lineBufRead + i) % LINE_BUF_SIZE;
            int lineIdx = lineBuf[lineBufIdx];
            float2 lb = make_float2(line_begins[lineIdx].x-pose.x, line_begins[lineIdx].y-pose.y);
            float2 le = make_float2(line_ends[lineIdx].x-pose.x, line_ends[lineIdx].y-pose.y);

            // 计算起始和终止栅格
            auto s_angle = vec_atan2_0_360(lb);
            auto e_angle = vec_atan2_0_360(le);
            int s_grid = s_angle * g_hparams.resolu_inv;
            int e_grid = e_angle * g_hparams.resolu_inv;

            // 计算相交射线数
            int frag = e_grid-s_grid;
            frag += (e_grid < s_grid) ? g_hparams.ray_num : 0;

            // 相交射线数 >0 时存入frLineFrag
            if(frag > 0) {
                frLineFrag[frBufWrite % FR_BUF_SIZE] = frag;
                frLineBuf[(frBufWrite++) % FR_BUF_SIZE] =
                    std::make_tuple(lineIdx, s_grid, e_grid);
            }
        }

        // printf("[SEPERATE] FRLINE VALID: %d\n", frBufWrite-frBufRead);

        // 发射
        while(frBufWrite-frBufRead > 0)
        {
            // decompress，加载512个frag
            // int oldfragRead = frBufRead;
            while(frBufRead != frBufWrite && fragWrite - fragRead < TOTAL_EMITION)
            {
                for(int j=0; j<frLineFrag[frBufRead]; j++)
                    fragBuf[fragWrite++ % FRAG_BUF_SIZE] = {frBufRead, j};
                frBufRead++;
            }
            // printf("[EMIT] LOAD LINE {%d-%d}, TOTAL_FRAG: %d \n", oldfragRead, frBufRead, fragWrite-fragRead);

            // 填充
            for(int i=0; i<TOTAL_EMITION; i+=4)
            {
                if(fragRead + i >= fragWrite) break;
                for(int j=0; j<EMIT_PER_THREAD; j++)
                {
                    int fragBufIdx = fragRead + i + j;
                    if(fragBufIdx >= fragWrite)
                        break;
                    const auto& [frLineIdx, frag] = fragBuf[fragBufIdx % FRAG_BUF_SIZE];
                    const auto& [lineIdx, s_grid, e_grid] = frLineBuf[frLineIdx];
                    float2 lb = make_float2(line_begins[lineIdx].x-pose.x, line_begins[lineIdx].y-pose.y);
                    float2 le = make_float2(line_ends[lineIdx].x-pose.x, line_ends[lineIdx].y-pose.y);
                    int grid = (s_grid + frag + 1) % g_hparams.ray_num;
                    response[grid] = min(response[grid], getR(lb, le, grid*g_hparams.resolu)*100);
                }
            }

            fragRead = min(fragRead+TOTAL_EMITION, fragWrite);
        }

        lineBufRead = min(lineBufRead+CTA_SIZE, lineBufWrite);

    // TODO. 深度预裁剪
    }

    return response;
}

template<typename T>
sf::Vector2<T> convertPoint(sf::RenderWindow &window, sf::Vector2<T> v)
{
    auto size = window.getView().getSize();
    return sf::Vector2<T>(v.x * SCALING + OFFSET, size.y - (v.y*SCALING + OFFSET));
}

template<typename T>
sf::Vector2<T> invConvertPoint(sf::RenderWindow &window, sf::Vector2<T> v)
{
    auto size = window.getSize();
    return sf::Vector2<T>((v.x-OFFSET)/(size.x/800.f) / SCALING, ((size.y-v.y)-OFFSET)/(size.y/600.f) /SCALING);
}


void draw(const std::vector<float2>& lbegins, const std::vector<float2>& lends) {
    sf::RenderWindow window(sf::VideoMode(800, 600), "SFML Draw Lines");
    window.setVerticalSyncEnabled(true); // call it once, after creating the window

    float2 startPoint = make_float2(2, 2); // 起点，初始为 {2, 2}
    std::vector<std::pair<float2, float2>> ray_shape;

    sf::Font font;
    if (!font.loadFromFile("/usr/share/fonts/truetype/dejavu/DejaVuSerif.ttf")) { // 确保有合适的字体文件
        std::cerr << "Failed to load font!" << std::endl;
        return;
    }

    // 文本对象
    sf::Text mousePositionText;
    mousePositionText.setFont(font);
    mousePositionText.setCharacterSize(20); // 设置字体大小
    mousePositionText.setFillColor(sf::Color::Green); // 设置文字颜色
    mousePositionText.setPosition(10, 10); // 设置文字位置

    // 主循环
    while (window.isOpen()) {
        sf::Event event;

        sf::Vector2i mousePos = sf::Mouse::getPosition(window);
        sf::Vector2f mouseWorldPos = invConvertPoint<float>(window, {mousePos.x, mousePos.y});

        while (window.pollEvent(event)) {
            if (event.type == sf::Event::Closed) {
                window.close();
            }

            // 鼠标点击事件
            if (event.type == sf::Event::MouseButtonPressed && event.mouseButton.button == sf::Mouse::Left) {
                // 获取鼠标点击位置


                // 转换为逻辑坐标
                startPoint.x = mouseWorldPos.x;
                startPoint.y = mouseWorldPos.y;

                // 打印鼠标点击位置（调试用）
                std::cout << "Mouse clicked at: (" << startPoint.x << ", " << startPoint.y << ")\n";

                ray_shape.clear();
                std::vector<float> rays = rasterGPU(
                    lbegins.size(),
                    make_float3(startPoint.x, startPoint.y, 0),
                    lbegins,
                    lends);
                for (size_t i = 0; i < rays.size(); i++) {
                    float angle = i * g_hparams.resolu;
                    float r = rays[i] / 100.f;
                    float2 endPoint = make_float2(r * cosf(angle) + startPoint.x, r * sinf(angle) + startPoint.y);
                    ray_shape.push_back({ startPoint, endPoint});
                }

            }
        }

        // 获取鼠标位置并更新文本内容
        mousePositionText.setString("Mouse: (" + std::to_string(mouseWorldPos.x) + ", " + std::to_string(mouseWorldPos.y) + ")");

        // 清空窗口
        window.clear(sf::Color::Black);

        // 绘制所有直线
        for (size_t i=0; i<lbegins.size(); i++) {
            // 创建一个 sf::VertexArray 用于绘制线段
            sf::VertexArray lineShape(sf::Lines, 2);

            // 设置第一个点
            lineShape[0].position = convertPoint(window, sf::Vector2f(lbegins[i].x, lbegins[i].y));
            lineShape[0].color = sf::Color::Red;

            // 设置第二个点
            lineShape[1].position = convertPoint(window, sf::Vector2f(lends[i].x, lends[i].y));
            lineShape[1].color = sf::Color::White;

            // 绘制线段
            window.draw(lineShape);
        }

        for (const auto& line : ray_shape) {
            // 创建一个 sf::VertexArray 用于绘制线段
            sf::VertexArray lineShape(sf::Lines, 2);

            // 设置第一个点
            lineShape[0].position = convertPoint(window, sf::Vector2f(line.first.x, line.first.y));
            lineShape[0].color = sf::Color::Blue;

            // 设置第二个点
            lineShape[1].position = convertPoint(window, sf::Vector2f(line.second.x, line.second.y));
            lineShape[1].color = sf::Color::White;

            // 绘制线段
            window.draw(lineShape);
        }

        // 绘制鼠标位置文本
        window.draw(mousePositionText);
        // 显示窗口内容
        window.display();
    }
}



int main()
{
    google::InstallFailureSignalHandler();

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_params), &g_hparams, sizeof(g_hparams)));

    auto map_generator = std::make_unique<map_gen::CellularAutomataGenerator>(MAP_WIDTH, MAP_HEIGHT);
    // auto map_generator = std::make_unique<MessyBSPGenerator>(MAP_WIDTH, MAP_HEIGHT);
    map_generator->generate();
    auto map = map_generator->getMap();
    auto shapes = map_gen::processGridmap(map, GRID_SIZE);

    std::vector<float2> lbegins, lends;
    std::for_each(shapes.begin(), shapes.end(), [&lbegins, &lends](const auto& polygons){
        for(size_t pgi=0; pgi<polygons.size(); pgi++) {
            auto pg = polygons[pgi];
            pg.push_back(pg.front());
            // if(polygons.size() != 2 || (polygons.size() == 2 && pgi == 1)) continue;
            for(size_t i=0; i<pg.size()-1; i++) {
                auto lb = make_float2(pg[i].x, pg[i].y);
                auto le = make_float2(pg[i+1].x, pg[i+1].y);
                // if(polygons.size() == 2)
                std::swap(lb, le);
                lbegins.push_back(lb);
                lends.push_back(le);
            }
        }

    });

    // lbegins =
    //     {
    //         {1,1}  ,
    //         {1,11} ,
    //         {11,11},
    //         {11,1} ,
    //     };

    // lends =
    //     {
    //         {1,11},
    //         {11,11},
    //         {11,1},
    //         {1,1},
    //     };

    draw(lbegins, lends);

    // std::copy(rays.begin(), rays.end(), std::ostream_iterator<float>(std::cout, ", "));
}