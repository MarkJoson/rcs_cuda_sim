#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
#include <hipblas.h>

#include "MincoTrajHelper.hh"
#include "component/MincoTrajMover.hh"
#include "core/EnvGroupManager.cuh"
#include "core/MessageBus.hh"
#include "core/SimulatorContext.hh"
#include "core/storage/GTensorConfig.hh"
#include "cuda_helper.h"
#include "geometry/GeometryManager.cuh"

// #include "MincoTrajHelper.hh"

using namespace cuda_simulator::core;

namespace cuda_simulator {
namespace minco_traj_mover {

#define CUBLAS_CHECK(err)                                                      \
  do {                                                                         \
    hipblasStatus_t err_ = (err);                                               \
    if (err_ != HIPBLAS_STATUS_SUCCESS) {                                       \
      std::printf("cublas error %d at %s:%d\n", err_, __FILE__, __LINE__);     \
      throw std::runtime_error("cublas error");                                \
    }                                                                          \
  } while (0)

constexpr int NUM_CKPTS = 4;

struct MincoTrajMover::Priv {
  hipblasHandle_t cublas_handle_;
  GTensor matF;      // 轨迹, 状态转移矩阵(6x6)
  GTensor matG;      // 目标位置, 输入矩阵(6x1)
  GTensor matC;      // 速度, 加速度, 检查矩阵(2xNUM_CKPTSx6),  [{vel/acc}, {ckpt0/1/2},
  GTensor matCmGinv; // 检查矩阵乘以输入矩阵(2xNUM_CKPTSx1)
  GTensor bound; // 2xNUM_CKPTSx3 [速度/加速度, ckpts, x/y/z方向]
};

MincoTrajMover::MincoTrajMover()
    : Component("minco_traj_mover"), priv_(std::make_unique<Priv>()) {

  std::vector<float> mat_f, mat_g, mat_ckpt;
  auto system = MincoTrajSystem(0.1, 0.2);
  // system.getMatF(&config.coeff_matF[0][0]);
  // system.getMatG(config.coeff_matG);
  // system.getMatCkpt(4, &(priv_->mat_ckpt[0][0][0]));

  // hipMemcpyToSymbol(HIP_SYMBOL(d_config), &config, sizeof(MincoMoverConfig));

  priv_->matCmGinv.reshape({2, NUM_CKPTS, 1});

  float max_vel_x = 1;
  float max_vel_y = 1;
  float max_vel_z = 1;
  float max_acc_x = 1;
  float max_acc_y = 1;
  float max_acc_z = 1;

  priv_->bound = GTensor::fromHostVector<float>(
      {max_vel_x, max_vel_y, max_vel_z, max_acc_x, max_acc_y, max_acc_z});

  // bound(2,3) reshape -> (2, 1, 3)
  priv_->bound = priv_->bound.reshape({2, 1, 3});
  // bound(2,1,3) * matCmGinv(2,NUMCKPT,1) => (2,NUMCKPT,3)
  priv_->bound *= priv_->matCmGinv;
  // bound(2,NUMCKPT,3) reshape -> (2,NUMCKPT,1,3)
  priv_->bound = priv_->bound.reshape({2, NUM_CKPTS, 1, 3});
}

MincoTrajMover::~MincoTrajMover() {
  CUBLAS_CHECK(hipblasDestroy(priv_->cublas_handle_));
}

// TODO. 与物理引擎的交互？

void MincoTrajMover::onNodeInit() {
  std::optional<Component::NodeOutputInfo> pose_info =
      getContext()->getOutputInfo("robot_entry", "pose");
  if (!pose_info.has_value()) {
    throw std::runtime_error("MincoTrajMover: robot_entry::pose not found.");
  }

  int num_robots = pose_info->shape[pose_info->shape.size() - 2];

  // 状态：位置x，位置y，角度z, [[coeff_x], [coeff_y], [coeff_z]]
  addState({"coeff", {num_robots, 3, 6}, NumericalDataType::kFloat32});

  // 输入：目标位置
  addInput({"posT", {num_robots, 3}, 0, ReduceMethod::STACK});
  // 输出：力
  addOutput({"force", {num_robots, 3}, NumericalDataType::kFloat32});
  // 输出：当前bound下的位置
  addOutput({"target_pos_bound",
             {
                 num_robots,
                 2,
             },
             NumericalDataType::kFloat32});
}

void MincoTrajMover::onNodeExecute(const core::NodeExecInputType &input,
                                   core::NodeExecOutputType &,
                                   core::NodeExecStateType &state) {
  float alpha = 1.0f;
  float beta = 0.0f;

  // 获取输入
  auto &coeff = *state.at("coeff");
  const auto &posT = *input.at("posT").front();

  int batch_count = coeff.elemCount() / 18;

  // 计算F*x, F(6x6), x(6xB3)
  GTensor FmX = GTensor::matmul(priv_->matF, coeff);

  // 计算G*u, G(6x1), u(1xB3)
  GTensor GmU = GTensor::matmul(priv_->matG, posT);

  // 计算V - Ckpt*FmX, [8,B3]=> [2(vel/acc),4(pts),B(batch_size),3(dim)],
  // 使用gemm运算; Ckpt(8x6), FmX(6xB3); m=8, n=batch_count * 3, k=6
  GTensor CkFmX = GTensor::matmul(priv_->matC, FmX);

  // 计算点除 CmGinv(2,NUMCKPT,1)->(2,NUMCKPT,1,1) * VmCkFmX(2,NUMCKPT,B,3) ->
  // (2,NUMCKPT,B,3)
  CkFmX *= priv_->matCmGinv.reshape({2, NUM_CKPTS, 1, 1});

  // 合并速度和加速度的ckpt维度，取最大值和最小值, (2*CKPT,B,3) => (B,3)
  auto max_bound =
      (priv_->bound - CkFmX).reshape({2 * NUM_CKPTS, batch_count, 3}).max(0);
  auto min_bound =
      (priv_->bound + CkFmX).reshape({2 * NUM_CKPTS, batch_count, 3}).min(0);

  // posT: (B,3)
  auto new_posT = posT.clamp(min_bound, max_bound);

  // 迭代计算新的coeff
  coeff = FmX + GTensor::matmul(priv_->matG, new_posT);
}

void MincoTrajMover::onNodeStart() {
  CUBLAS_CHECK(hipblasCreate(&priv_->cublas_handle_));
}

void MincoTrajMover::onNodeReset(const core::GTensor &reset_flags,
                                 core::NodeExecStateType &state) {
}

void MincoTrajMover::onEnvironGroupInit() {
}

} // namespace minco_traj_mover
} // namespace cuda_simulator
