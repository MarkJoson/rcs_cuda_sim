#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>

#include "core/storage/Scalar.hh"
#include "cuda_helper.h"

#include "component/LidarSensor.hh"
#include "core/Component.hh"
#include "core/EnvGroupManager.cuh"
#include "core/SimulatorContext.hh"
#include "core/core_types.hh"
#include "geometry/GeometryManager.cuh"
#include "geometry/shapes.hh"

using namespace cuda_simulator::core;

namespace cuda_simulator {
namespace lidar_sensor {

#define CTA_SIZE (128)
#define RASTER_WARPS (CTA_SIZE / 32)

#define LINE_BUF_SIZE (CTA_SIZE * 2)
#define FR_BUF_SIZE (CTA_SIZE * 2)
#define FRAG_BUF_SIZE (CTA_SIZE * 6)

#define EMIT_PER_THREAD (2)
#define TOTAL_EMITION (EMIT_PER_THREAD * CTA_SIZE)

#define LIDAR_LINES_LOG2 (4) // 128lines
#define LIDAR_LINES (1 << LIDAR_LINES_LOG2)

static constexpr float LIDAR_MAX_RANGE = 8.0f;
static constexpr float LIDAR_RESOLU = ((2 * M_PI) / LIDAR_LINES);
static constexpr float LIDAR_RESOLU_INV = (LIDAR_LINES / (2 * M_PI));
static constexpr uint32_t LIDAR_CVT_U16_SCALE = 1024;

static constexpr uint32_t LIDAR_MAX_RESPONSE = (uint32_t)(LIDAR_MAX_RANGE * LIDAR_CVT_U16_SCALE) << 16;

__host__ __device__ bool lineVisibleCheck(float2 vs, float2 ve, float max_range) {
  // 粗过滤可见边，使用距离+朝向的判定

  // !假定多边形按照逆时针排列
  float cdot = vs.x * ve.y - vs.y * ve.x;

  float dx = ve.x - vs.x;
  float dy = ve.y - vs.y;
  // float len = sqrtf(dx*dx+dy*dy);
  float invsqrt = rsqrtf(dx * dx + dy * dy);
  float dist = fabs(cdot) * invsqrt;

  return dist < max_range && cdot < 0;
}

__forceinline__ __host__ __device__ float vec_atan2_0_360(float2 vec) {
  float angle = atan2f(vec.y, vec.x);
  return angle < 0 ? angle + 2 * M_PI : angle;
}

__forceinline__ __host__ __device__ float getR(float2 lb, float2 le, float theta) {
  float sin, cos;
  sincosf(theta, &sin, &cos);
  float2 dvec = make_float2(le.x - lb.x, le.y - lb.y);
  return (lb.y * dvec.x - lb.x * dvec.y) / (dvec.x * sin - dvec.y * cos);
}

__forceinline__ __host__ __device__ uint32_t genMask(int start, int end) {
  return ((1 << end) - 1) & ~((1 << start) - 1);
  // int pop = __builtin_popcount(mask);
}

__forceinline__ __device__ float4 readLine(int num_dyn_lines, const float4 *__restrict__ static_lines,
                                           const float4 *__restrict__ dyn_lines, int lineIdx) {

  // 动态线段的数量固定，先处理动态线段
  if (lineIdx < num_dyn_lines)
    return dyn_lines[lineIdx]; // dyn_lines: [group, env, lines, 4]
  else
    return static_lines[lineIdx - num_dyn_lines];
}

__global__ void rasterKernel(const ConstantMemoryVector<uint32_t> num_static_lines, // 每个场景中的静态线段数量
                             const float4 *__restrict__ static_lines,               // 线段的起点
                             int num_dyn_lines,                    // 场景中的动态线段数量，所有场景统一
                             const float4 *__restrict__ dyn_lines, // 动态线段数组
                             const float4 *__restrict__ poses,     // 机器人的位姿
                             uint32_t *__restrict__ lidar_response // 激光雷达的响应
) {
  using BlockScan = hipcub::BlockScan<uint32_t, CTA_SIZE>;
  using BlockRunLengthDecodeT = hipcub::BlockRunLengthDecode<uint32_t, CTA_SIZE, 1, EMIT_PER_THREAD>;

  volatile __shared__ uint32_t s_lineBuf[LINE_BUF_SIZE];           // 1K
  volatile __shared__ uint32_t s_frLineIdxBuf[FR_BUF_SIZE];        // 1K
  volatile __shared__ uint32_t s_frLineSGridFragsBuf[FR_BUF_SIZE]; // 1K
  __shared__ uint32_t s_lidarResponse[LIDAR_LINES];                // 256B
  __shared__ union {
    typename BlockScan::TempStorage scan_temp_storage;
    typename BlockRunLengthDecodeT::TempStorage decode_temp_storage;
  } temp_storage;

  /// Grid: (num_lidars, num_envs, num_groups)
  //  Block: (CTA_SIZE, 1, 1)
  int group_id = blockIdx.z;
  int env_inst_id = blockIdx.z * gridDim.y + blockIdx.y;
  int lidar_inst_id = env_inst_id * gridDim.x + blockIdx.x;

  // 每个场景有其对应的动态线段（由所有动态物体的位姿计算）
  const float4 *__restrict__ dyn_lines_in_env = env_inst_id * num_dyn_lines + dyn_lines;
  // 每个场景组有对应的静态线段
  const float4 *__restrict__ static_lines_in_group = static_lines + group_id;
  // 每个场景组静态线段的数量不同
  int num_static_line_in_group = num_static_lines[group_id];
  // dynamic line在前，static line在后
  int num_lines = num_dyn_lines + num_static_line_in_group;

  uint32_t tid = threadIdx.x;
  uint32_t totalLineRead = 0;
  uint32_t lineBufRead = 0, lineBufWrite = 0;
  uint32_t frLineBufRead = 0, frLineBufWrite = 0;
  float4 pose = poses[lidar_inst_id];

  /********* 初始化lidar数据 *********/
  for (int i = tid; i < LIDAR_LINES; i += CTA_SIZE)
    s_lidarResponse[i] = LIDAR_MAX_RESPONSE;

  // if(tid == 0) printf("---------##### Task RECEIVED\n");

  for (;;) {
    /********* Load Lines to Buffer *********/
    while (lineBufWrite - lineBufRead < CTA_SIZE && totalLineRead < num_lines) {
      // if(tid == 0) printf("[READ] LINE RANGE: %d~%d.\n", totalLineRead, totalLineRead+CTA_SIZE);
      uint32_t visibility = false;
      int lineIdx = totalLineRead + tid;
      if (lineIdx < num_lines) {
        // float4 line = lines[lineIdx];

        float4 line = readLine(num_dyn_lines, static_lines_in_group, dyn_lines_in_env, lineIdx);
        float2 lb = make_float2(line.x - pose.x, line.y - pose.y);
        float2 le = make_float2(line.z - pose.x, line.w - pose.y);
        visibility = lineVisibleCheck(lb, le, LIDAR_MAX_RANGE);
        // if(visibility)
        // {
        //     const auto &vs=lb, &ve=le;
        //     float cdot = vs.x*ve.y - vs.y*ve.x;
        //     float dx = ve.x-vs.x;
        //     float dy = ve.y-vs.y;
        //     float len = sqrtf(dx*dx+dy*dy);
        //     float dist = fabs(cdot)/len;
        //     printf("[READ] PASS:%03d, \t[%.2f,%.2f]->[%.2f,%.2f], \t[%.2f,%.2f]->[%.2f,%.2f], \tarea:%.2f,
        //     \tdist%.2f\n", lineIdx,
        //         line_begins[lineIdx].x, line_begins[lineIdx].y, line_ends[lineIdx].x, line_ends[lineIdx].y,
        //         lb.x, lb.y, le.x, le.y, cdot, dist);
        // }
      }

      uint32_t scan, scan_reduce;
      BlockScan(temp_storage.scan_temp_storage).ExclusiveSum(visibility, scan, scan_reduce);

      if (visibility) {
        s_lineBuf[lineBufWrite + scan] = lineIdx;
      }

      lineBufWrite += scan_reduce;

      totalLineRead += CTA_SIZE;
      __syncthreads();
    }

    // 第二部分继续的条件：已经读取了128个，或没有读取128个，但没有新的线段

    // if(tid == 0) printf("[READ] FINISHED! TOTAL READ: %d, LINE BUF:%d\n", totalLineRead, lineBufWrite);

    /********* 计算终止栅格，细光栅化，存入待发射线段缓冲区 *********/
    // do
    {
      int lineIdx = -1;
      int frag = 0;
      int e_grid = -1;
      if (lineBufRead + tid < lineBufWrite) {
        lineIdx = s_lineBuf[(lineBufRead + tid) % LINE_BUF_SIZE];

        float4 line = readLine(num_dyn_lines, static_lines_in_group, dyn_lines_in_env, lineIdx);
        float2 lb = make_float2(line.x - pose.x, line.y - pose.y);
        float2 le = make_float2(line.z - pose.x, line.w - pose.y);

        auto s_angle = vec_atan2_0_360(lb);
        auto e_angle = vec_atan2_0_360(le);
        int s_grid = s_angle * LIDAR_RESOLU_INV;
        e_grid = e_angle * LIDAR_RESOLU_INV;
        // 计算角度时，较大的角度减较小的角度，这正好与逆时针的定义相反, 因此是起点-终点
        frag = (s_grid - e_grid) + ((s_grid < e_grid) ? LIDAR_LINES : 0);
        // printf("[RASTER] THREAD:%d, LINE_ID:%d, SGRID:%d, EGRID:%d, FRAG:%d\n", tid, lineIdx, s_grid, e_grid, frag);
      }

      // 压缩到FR_BUF队列中
      uint32_t scan, scan_sum;
      BlockScan(temp_storage.scan_temp_storage).ExclusiveSum(frag > 0, scan, scan_sum);
      if (frag > 0) {
        uint32_t idx = (frLineBufWrite + scan) % FR_BUF_SIZE;
        s_frLineIdxBuf[idx] = lineIdx;
        // 这里同理，需要使用e_grid而不是s_grid
        s_frLineSGridFragsBuf[idx] = (e_grid << 16) | (frag & 0xffff);
      }
      frLineBufWrite += scan_sum;
      __syncthreads();

      //
      lineBufRead = min(lineBufRead + CTA_SIZE, lineBufWrite);
    }

    // 此时要么 读取了128个，要么 lineBuf处理完了
    // if(tid == 0) printf("[RASTER] FRBuf:[R:%d,W:%d] VALID: %d\n", frLineBufRead, frLineBufWrite, frLineBufWrite -
    // frLineBufRead);

    // 第三部分继续的条件：读取到128个，或未读取到128个，但是已经无法再读取新的线段；
    if (frLineBufWrite - frLineBufRead < CTA_SIZE && (lineBufRead < lineBufWrite || totalLineRead < num_lines)) {
      // if(tid == 0) printf("[RASTER] CONTINUE LOOP!\n");
      continue;
    }

    // if(tid == 0) printf("[RASTER] FINISHED!\n");

    /********* Count and Emit *********/
    do {
      // if(tid == 0) printf("[EMIT] LOAD LINE [%d-%d]\n", frLineBufRead, frLineBufWrite);
      // 加载CTA_SIZE个到缓冲区，准备进行Decode
      uint32_t runValue[1] = {0}, runLength[1] = {0};
      int frLineBufIdx = frLineBufRead + tid;
      if (frLineBufIdx < frLineBufWrite) {
        frLineBufIdx = frLineBufIdx % FR_BUF_SIZE;
        runValue[0] = frLineBufIdx;
        runLength[0] = s_frLineSGridFragsBuf[frLineBufIdx] & 0xffff; // 取低16位的frag
      }
      frLineBufRead = min(frLineBufRead + CTA_SIZE, frLineBufWrite);
      __syncthreads();

      uint32_t total_decoded_size = 0;
      BlockRunLengthDecodeT blk_rld(temp_storage.decode_temp_storage, runValue, runLength, total_decoded_size);

      // 将本次读取的 CTA_SIZE*EMIT_PER_LINE 个frag全部发射
      uint32_t decoded_window_offset = 0;
      while (decoded_window_offset < total_decoded_size) {
        uint32_t relative_offsets[2];
        uint32_t decoded_items[2];
        uint32_t num_valid_items = min(total_decoded_size - decoded_window_offset, CTA_SIZE * EMIT_PER_THREAD);
        blk_rld.RunLengthDecode(decoded_items, relative_offsets, decoded_window_offset);
        decoded_window_offset += num_valid_items;

#pragma unroll
        for (int i = 0; i < 2; i++) {
          if (tid * EMIT_PER_THREAD + i >= num_valid_items)
            break;

          int fragIdx = relative_offsets[i];
          uint32_t frLineBufIdx = decoded_items[i];
          uint32_t lineIdx = s_frLineIdxBuf[frLineBufIdx];
          int e_grid = s_frLineSGridFragsBuf[frLineBufIdx] >> 16;

          float4 line = readLine(num_dyn_lines, static_lines_in_group, dyn_lines_in_env, lineIdx);
          float2 lb = make_float2(line.x - pose.x, line.y - pose.y);
          float2 le = make_float2(line.z - pose.x, line.w - pose.y);

          int grid = (e_grid + fragIdx + 1) % LIDAR_LINES;
          uint16_t resp_u16 = getR(lb, le, grid * LIDAR_RESOLU) * 1024; // 10位定点小数表示，最大距离64m, 分辨率: 0.0625m
          uint32_t resp_idx = resp_u16 << 16 | lineIdx & 0xffff;
          atomicMin_block(&s_lidarResponse[grid], resp_idx);
        }
      }
      __syncthreads();
    } while (frLineBufWrite != frLineBufRead &&
             totalLineRead >=
                 num_lines); // 继续的条件：已经没有办法读取更多的frag线段，则需要将剩余的frlineBufWrite处理完

    // if(tid == 0) printf("[EMIT] FINISHED!\n");

    // 全部线段已经处理完
    if (totalLineRead >= num_lines)
      break;
  }

  for (int i = tid; i < LIDAR_LINES; i += CTA_SIZE)
    lidar_response[i] = s_lidarResponse[i];
}

void LidarSensor::onNodeReset(const TensorHandle &reset_flags, NodeExecStateType &state) {
  // 重置LidarSensor
}

void LidarSensor::onEnvironGroupInit() {
  // 初始化LidarSensor
  getGeometryManager()->createStaticPolyObj(0,
                                            geometry::SimplePolyShapeDef({
                                                {1.0, 0.0},
                                                {1.0, 1.0},
                                                {0.0, 1.0},
                                                {0.0, 0.0},
                                            }),
                                            {{2, 0}, 0});
  getGeometryManager()->createStaticPolyObj(0,
                                            geometry::SimplePolyShapeDef({
                                                {1.0, 0.0},
                                                {1.0, 1.0},
                                                {0.0, 1.0},
                                                {0.0, 0.0},
                                            }),
                                            {{4, 2}, 0});
  getGeometryManager()->createStaticPolyObj(0,
                                            geometry::SimplePolyShapeDef({
                                                {1.0, 0.0},
                                                {1.0, 1.0},
                                                {0.0, 1.0},
                                                {0.0, 0.0},
                                            }),
                                            {{2, 4}, 0});
  getGeometryManager()->createStaticPolyObj(0,
                                            geometry::SimplePolyShapeDef({
                                                {1.0, 0.0},
                                                {1.0, 1.0},
                                                {0.0, 1.0},
                                                {0.0, 0.0},
                                            }),
                                            {{4, 0}, 0});
}

LidarSensor::LidarSensor() : Component("lidar_sensor") {
  // addDependence({"map_generator"});
  addDependence({"robot_entry"});
}

void LidarSensor::onNodeInit() {
  // [group, env, inst, 4]
  std::optional<Component::NodeOutputInfo> pose_info = getContext()->getOutputInfo("robot_entry", "pose");
  if (!pose_info.has_value()) {
    throw std::runtime_error("LidarSensor: robot_entry::pose not found.");
  }
  MessageShapeRef input_shape(pose_info.value().shape);
  num_inst_ = input_shape[input_shape.size() - 2];
  input_shape.copyTo(output_shape_);
  output_shape_[output_shape_.size() - 1] = LIDAR_LINES;

  addInput({"pose", input_shape, 0, ReduceMethod::STACK});

  addOutput({"lidar", output_shape_, NumericalDataType::kUInt32});
}

void LidarSensor::onNodeExecute(const NodeExecInputType &input, NodeExecOutputType &output) {

  /// Grid: (num_lidars, num_envs, num_groups)
  //  Block: (CTA_SIZE, 1, 1)
  dim3 block_dim{CTA_SIZE, 1, 1};
  uint32_t num_group = getEnvGroupMgr()->getNumActiveGroup();
  uint32_t num_envs = getEnvGroupMgr()->getNumEnvPerGroup();
  dim3 grid_dim{num_inst_, num_envs, num_group};

  uint32_t num_dyn_lines = getGeometryManager()->getNumDynLines();

  const float4 *dyn_lines = getGeometryManager()->getDynamicLines().typed_data<float4>();
  const float4 *static_lines = getGeometryManager()->getStaticLinesDeviceTensor().typed_data<float4>();
  const float4 *pose = input.at("pose").begin()->typed_data<float4>();
  const ConstantMemoryVector<uint32_t> &num_static_lines = getGeometryManager()->getNumStaticLines()->getDeviceData();

  uint32_t *lidar = output.at("lidar").typed_data<uint32_t>();

  rasterKernel<<<grid_dim, block_dim>>>(num_static_lines, static_lines, num_dyn_lines, dyn_lines, pose, lidar);

  checkCudaErrors(hipDeviceSynchronize());
  // TODO. 删除自己的dynamic_line

  // std::cout << "LidarSensor: "<< output.at("lidar")/65536/1024.f << std::endl;

  // rasterKernel, block大小 == 128, 1个block处理1个机器人. grid大小 ==
  // (环境组数,环境数,机器人数) rasterKernel: Input:[poses],
  // Output:[lidar_response] 线段数据：储存在全局内存，numLines, 储存在
  // constant 内存
  // 1. 从场景管理器中获得所有线段的数据：line_begins, line_ends
  // 2. 计算机器人的位姿poses
  // 3. 发布激光雷达的响应lidar_response
}

float LidarSensor::getLidarRange() const {
  return LIDAR_MAX_RANGE;
}

float LidarSensor::getLidarResolution() const {
  return LIDAR_RESOLU;
}

float LidarSensor::getLidarRayNum() const {
  return LIDAR_LINES;
}

} // namespace lidar_sensor
} // namespace cuda_simulator
